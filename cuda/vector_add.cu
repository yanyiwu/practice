#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vectors A and B
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vectors A and B
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // Free device global memory
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    // Free host memory
    if (h_A) free(h_A);
    if (h_B) free(h_B);
    if (h_C) free(h_C);

    printf("Done\n");
    return 0;
} 